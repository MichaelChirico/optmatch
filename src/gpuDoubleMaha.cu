#include "hip/hip_runtime.h"
#include<R.h>

#define NTHREADS 64

void safeCudaMallocDouble(double ** vect, int ndoubles) {
	hipError_t err = hipSuccess;
	err = hipMalloc((void **) vect, ndoubles * sizeof(double));
	if(err != hipSuccess) error(hipGetErrorString(err));
}

size_t safeCudaMallocPitchDouble(double ** vect, int width, int height) {
	hipError_t err = hipSuccess;
	size_t pitch;
	err = hipMallocPitch(vect, &pitch, width * sizeof(double), height);
	if(err != hipSuccess) error(hipGetErrorString(err));
	return pitch;
}

void safeToDeviceDouble(double * a, const double * b, int ndoubles) {
	hipError_t err = hipSuccess;
	err = hipMemcpy(a, b, ndoubles * sizeof(double), hipMemcpyHostToDevice);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void safeToDevice2DDouble(double * a, size_t aPitch,
	const double * b, size_t nrows, size_t ncols)
{
	hipError_t err = hipSuccess;
	err = hipMemcpy2D(a, aPitch, b, ncols * sizeof(double),
		ncols * sizeof(double), nrows, hipMemcpyHostToDevice);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void safeFromDeviceDouble(double * a, const double * b, int ndoubles) {
	hipError_t err = hipSuccess;
	err = hipMemcpy(a, b, ndoubles * sizeof(double), hipMemcpyDeviceToHost);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void checkCudaErrorDouble() {
	hipError_t err = hipSuccess;
	hipGetLastError();
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void safeCudaFreeDouble(double * a) {
	hipError_t err = hipSuccess;
	err = hipFree(a);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

__global__ void dDoubleMaha(int nvectors, int n,
	const double * vectors1, size_t v1Pitch,
	const double * vectors2, size_t v2Pitch,
	const double * mat, size_t matPitch,
	double * result)
{
	int k, i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nvectors) return;

	double
		sum = 0, innerSum;
	const double
		* matCol,
		* v1i = vectors1 + i * v1Pitch,
		* v2i = vectors2 + i * v2Pitch;

	for(int j = 0; j < n; j++) {
		innerSum = 0;
		matCol = mat + j * matPitch;
		for(k = 0; k < n; k++)
			innerSum += (v1i[k] - v2i[k]) * matCol[k];
		sum += innerSum * (v1i[j] - v2i[j]);
	}
	result[i] = sqrt(sum);
}

extern "C"
void gpuDoubleMaha(const int * vectorSetSize, const int * vectorLength,
	const double * vectorSet1, const double * vectorSet2, const double * mat,
	double * result)
{
	int
		nv = *vectorSetSize, n = *vectorLength;

	double
		* dMat = NULL, * dVectorSet1 = NULL, * dVectorSet2 = NULL,
		* dResult = NULL;
	size_t
		v1Pitch, v2Pitch, matPitch;

	safeCudaMallocDouble(&dResult, nv);

	v1Pitch = safeCudaMallocPitchDouble(&dVectorSet1, n, nv);
	v2Pitch = safeCudaMallocPitchDouble(&dVectorSet2, n, nv);
	matPitch = safeCudaMallocPitchDouble(&dMat, n, nv);

	safeToDevice2DDouble(dVectorSet1, v1Pitch, vectorSet1, nv, n);
	safeToDevice2DDouble(dVectorSet2, v2Pitch, vectorSet2, nv, n);
	safeToDevice2DDouble(dMat, matPitch, mat, n, n);

	size_t nblocks = ceil((double) nv /(double) NTHREADS);

	dDoubleMaha<<<nblocks, NTHREADS>>>(nv, n,
		dVectorSet1, v1Pitch / sizeof(double),
		dVectorSet2, v2Pitch / sizeof(double),
		dMat, matPitch / sizeof(double), dResult);
	checkCudaErrorDouble();
	safeFromDeviceDouble(result, dResult, nv);

	safeCudaFreeDouble(dVectorSet1);
	safeCudaFreeDouble(dVectorSet2);
	safeCudaFreeDouble(dMat);
	safeCudaFreeDouble(dResult);
}
