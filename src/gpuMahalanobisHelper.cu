
#include <hip/hip_runtime.h>
#define NTHREADS 16

// computes the dot product of two vectors u and v of length n

__device__ float dDotProduct(int n, const float * u, const float * v) {
	float sum = 0.f;
	for(int i = 0; i < n; i++)
		sum += u[i] * v[i];
	return sum;
}

__global__ void dMahalanobisHelper(int vectorSetSize, int vectorLength,
	const float * vectorSet1, const float * vectorSet2, const float * mat,
	float * temp1, float * temp2, float * result)
{
	int
		i = blockDim.x * blockIdx.x + threadIdx.x,
		nv = vectorSetSize, n = vectorLength;

	if(i >= vectorSetSize) return;

	float
		* temp1i = temp1 + i * vectorLength,
		* temp2i = temp2 + i * vectorLength;

	for(int j = 0; j < vectorLength; j++)
		temp1i[j] = vectorSet1[i + j * nv] - vectorSet2[i + j * nv];
	for(int j = 0; j < vectorLength; j++)
		temp2i[j] = dDotProduct(n, temp1i, mat + j * n);

	result[i] = sqrtf( dDotProduct(n, temp1i, temp2i) );
}

// computes sqrt((u_i - v_i) * m * (u_i - v_i)) for two sets of vectors
// u and v and a matrix m. The matrix m should have dim n x n where each
// u_i and v_i are of length n. This is intended to fill in the last part of a
// mahabalonis distance calculation for a set of vectors u and v,
// one set a treatment and the other a control.

// The matrix m is expected to be stored column major in memory
// the ith column's elements are stored consecutively at m + i * n
// the jth row will be at m + 0 * n + j, m + n + j, m + 2 * n + j, ...

// each vector is expected to be a row vector of length n, with the whole
// set stored as an nv x n matrix w in column major order :(
// to get at the jth vector in the set, you must index the elements as
// w + 0 * nv + j, w + nv + j, w + 2 * nv + j, ..., w + (n - 1) * nv + j

void gpuMahalanobisHelper(const int * vectorSetSize, const int * vectorLength,
	const float * vectorSet1, const float * vectorSet2, const float * mat,
	float * result)
{
	int
		nv = *vectorSetSize, n = *vectorLength;

	float
		* dVectorSet1, * dVectorSet2, * dMat,
		* temp1, * temp2, * dResult;

	size_t
		resultBytes = nv * sizeof(float),
		matBytes = n * n * sizeof(float),
		vectorSetBytes = nv * n * sizeof(float);

	hipMalloc((void **) & dVectorSet1, vectorSetBytes);
	hipMalloc((void **) & dVectorSet2, vectorSetBytes);
	hipMalloc((void **) & dMat, matBytes);
	hipMalloc((void **) & temp1, vectorSetBytes);
	hipMalloc((void **) & temp2, vectorSetBytes);
	hipMalloc((void **) & dResult, resultBytes);

	hipMemcpy(dVectorSet1, vectorSet1, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dVectorSet2, vectorSet2, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dMat, mat, matBytes, hipMemcpyHostToDevice);

	dim3 dimBlock(NTHREADS, 1, 1);
	int gx = ceil((double) nv /(double) dimBlock.x);
	dim3 dimGrid(gx, 1, 1);

	dMahalanobisHelper<<<dimBlock, dimGrid>>>(nv, n, dVectorSet1, dVectorSet2,
		dMat, temp1, temp2, dResult);

	hipFree(temp1);
	hipFree(temp2);
	hipFree(dVectorSet1);
	hipFree(dVectorSet2);
	hipFree(dMat);

	hipMemcpy(result, dResult, resultBytes, hipMemcpyDeviceToHost);
	hipFree(dResult);
}
