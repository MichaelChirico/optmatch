#include "hip/hip_runtime.h"
#include<R.h>

#define NTHREADS 256

texture<float, hipTextureType2D, hipReadModeElementType> dMat;

void safeCudaMallocFloat(float ** vect, int nfloats) {
	hipError_t err = hipSuccess;
	err = hipMalloc((void **) vect, nfloats * sizeof(float));
	if(err != hipSuccess) error(hipGetErrorString(err));
}

size_t safeCudaMallocPitchFloat(float ** vect, int width, int height) {
	hipError_t err = hipSuccess;
	size_t pitch;
	err = hipMallocPitch(vect, &pitch, width * sizeof(float), height);
	if(err != hipSuccess) error(hipGetErrorString(err));
	return pitch;
}

void safeToDeviceFloat(float * a, const float * b, int nfloats) {
	hipError_t err = hipSuccess;
	err = hipMemcpy(a, b, nfloats * sizeof(float), hipMemcpyHostToDevice);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void safeToDevice2DFloat(float * a, size_t aPitch,
	const float * b, size_t nrows, size_t ncols)
{
	hipError_t err = hipSuccess;
	err = hipMemcpy2D(a, aPitch, b, ncols * sizeof(float),
		ncols * sizeof(float), nrows, hipMemcpyHostToDevice);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void safeFromDeviceFloat(float * a, const float * b, int nfloats) {
	hipError_t err = hipSuccess;
	err = hipMemcpy(a, b, nfloats * sizeof(float), hipMemcpyDeviceToHost);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void checkCudaError() {
	hipError_t err = hipSuccess;
	hipGetLastError();
	if(err != hipSuccess) error(hipGetErrorString(err));
}

void safeCudaFree(float * a) {
	hipError_t err = hipSuccess;
	err = hipFree(a);
	if(err != hipSuccess) error(hipGetErrorString(err));
}

__global__ void dMahalanobisHelper(int nvectors, int n,
	const float * vectors1, size_t v1Pitch,
	const float * vectors2, size_t v2Pitch,
	float * result)
{
	int k, i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nvectors) return;

	float
		sum = 0, innerSum;
	const float
		* v1i = vectors1 + i * v1Pitch,
		* v2i = vectors2 + i * v2Pitch;

	for(int j = 0; j < n; j++) {
		innerSum = 0;
		for(k = 0; k < n; k++)
			innerSum += (v1i[k] - v2i[k]) * tex2D(dMat, j, k);
		sum += innerSum * (v1i[j] - v2i[j]);
	}
	result[i] = sqrtf(sum);
}

extern "C"
void gpuMahalanobisHelper(const int * vectorSetSize, const int * vectorLength,
	const float * vectorSet1, const float * vectorSet2, const float * mat,
	float * result)
{
	int
		nv = *vectorSetSize, n = *vectorLength;

	float
		* dVectorSet1 = NULL, * dVectorSet2 = NULL, * dResult = NULL;
	size_t v1Pitch, v2Pitch;

	// safeCudaMallocFloat(&dVectorSet1, nv * n);
	// safeCudaMallocFloat(&dVectorSet2, nv * n);

	safeCudaMallocFloat(&dResult, nv);

	v1Pitch = safeCudaMallocPitchFloat(&dVectorSet1, n, nv);
	v2Pitch = safeCudaMallocPitchFloat(&dVectorSet2, n, nv);

	safeToDevice2DFloat(dVectorSet1, v1Pitch, vectorSet1, nv, n);
	safeToDevice2DFloat(dVectorSet2, v2Pitch, vectorSet2, nv, n);

	// create mat in device memory as a read only 2D texture
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindFloat);
	hipArray * cuArray;
	hipMallocArray(&cuArray, &channelDesc, n, n);
	hipMemcpyToArray(cuArray, 0, 0, mat, n * n * sizeof(float),
		hipMemcpyHostToDevice);
	
	dMat.addressMode[0] = hipAddressModeClamp;
	dMat.addressMode[1] = hipAddressModeClamp;
	dMat.filterMode = hipFilterModePoint;
	dMat.normalized = false;

	hipBindTextureToArray(dMat, cuArray, channelDesc);
	// end create mat

	size_t nblocks = ceil((double) nv /(double) NTHREADS);

	dMahalanobisHelper<<<nblocks, NTHREADS>>>(nv, n,
		dVectorSet1, v1Pitch / sizeof(float),
		dVectorSet2, v2Pitch / sizeof(float),
		dResult);
	checkCudaError();
	safeFromDeviceFloat(result, dResult, nv);

	hipFreeArray(cuArray);
	safeCudaFree(dVectorSet1);
	safeCudaFree(dVectorSet2);
	// safeCudaFree(dMat);
	safeCudaFree(dResult);
}
