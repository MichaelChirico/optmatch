
#include <hip/hip_runtime.h>
#define NTHREADS 256

__global__ void dMahalanobisHelper(int nvectors, int n,
	const float * vectors1, const float * vectors2, const float * mat,
	float * result)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nvectors) return;

	int
		k, col, innerCol;
	float
		sum = 0, innerSum;

	for(int j = 0; j < n; j++) {
		innerSum = 0;
		col = j * n;
		for(k = 0; k < n; k++) {
			innerCol = i + k * nvectors;
			innerSum += (vectors1[innerCol] - vectors2[innerCol])
				* mat[k + col];
		}
		col = i + j * nvectors;
		sum += innerSum * (vectors1[col] - vectors2[col]);
	}
	result[i] = sqrtf(sum);
}

// computes sqrt((u_i - v_i) * m * (u_i - v_i)) for two sets of vectors
// u and v and a matrix m. The matrix m should have dim n x n where each
// u_i and v_i are of length n. This is intended to fill in the last part of a
// mahabalonis distance calculation for a set of vectors u and v,
// one set a treatment and the other a control.

// The matrix m is expected to be stored column major in memory
// the ith column's elements are stored consecutively at m + i * n
// the jth row will be at m + 0 * n + j, m + n + j, m + 2 * n + j, ...

// each vector is expected to be a row vector of length n, with the whole
// set stored as an nv x n matrix w in column major order :(
// to get at the jth vector in the set, you must index the elements as
// w + 0 * nv + j, w + nv + j, w + 2 * nv + j, ..., w + (n - 1) * nv + j

extern "C"
void gpuMahalanobisHelper(const int * vectorSetSize, const int * vectorLength,
	const float * vectorSet1, const float * vectorSet2, const float * mat,
	float * result)
{
	int
		nv = *vectorSetSize, n = *vectorLength;

	float
		* dVectorSet1, * dVectorSet2, * dMat, * dResult;

	size_t
		resultBytes = nv * sizeof(float),
		matBytes = n * n * sizeof(float),
		vectorSetBytes = nv * n * sizeof(float);

	hipMalloc((void **) & dVectorSet1, vectorSetBytes);
	hipMalloc((void **) & dVectorSet2, vectorSetBytes);
	hipMalloc((void **) & dMat, matBytes);
	hipMalloc((void **) & dResult, resultBytes);

	hipMemcpy(dVectorSet1, vectorSet1, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dVectorSet2, vectorSet2, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dMat, mat, matBytes, hipMemcpyHostToDevice);

	dim3 dimBlock(NTHREADS, 1, 1);
	int gx = ceil((double) nv /(double) dimBlock.x);
	dim3 dimGrid(gx, 1, 1);

	hipDeviceSynchronize();
	dMahalanobisHelper<<<dimBlock, dimGrid>>>(nv, n, dVectorSet1, dVectorSet2,
		dMat, dResult);
	hipDeviceSynchronize();

	hipMemcpy(result, dResult, resultBytes, hipMemcpyDeviceToHost);

	hipFree(dVectorSet1);
	hipFree(dVectorSet2);
	hipFree(dMat);
	hipFree(dResult);
}

__device__ double ddDotProduct(int n, const double * u, const double * v) {
	double sum = 0.0;
	for(int i = 0; i < n; i++)
		sum += u[i] * v[i];
	return sum;
}

__global__ void ddMahalanobisHelper(int vectorSetSize, int vectorLength,
	const double * vectorSet1, const double * vectorSet2, const double * mat,
	double * temp1, double * temp2, double * result)
{
	int
		i = blockDim.x * blockIdx.x + threadIdx.x,
		nv = vectorSetSize, n = vectorLength;

	if(i >= vectorSetSize) return;

	double
		* temp1i = temp1 + i * vectorLength,
		* temp2i = temp2 + i * vectorLength;

	for(int j = 0; j < vectorLength; j++)
		temp1i[j] = vectorSet1[i + j * nv] - vectorSet2[i + j * nv];
	for(int j = 0; j < vectorLength; j++)
		temp2i[j] = ddDotProduct(n, temp1i, mat + j * n);

	result[i] = sqrtf( ddDotProduct(n, temp1i, temp2i) );
}

// computes sqrt((u_i - v_i) * m * (u_i - v_i)) for two sets of vectors
// u and v and a matrix m. The matrix m should have dim n x n where each
// u_i and v_i are of length n. This is intended to fill in the last part of a
// mahabalonis distance calculation for a set of vectors u and v,
// one set a treatment and the other a control.

// The matrix m is expected to be stored column major in memory
// the ith column's elements are stored consecutively at m + i * n
// the jth row will be at m + 0 * n + j, m + n + j, m + 2 * n + j, ...

// each vector is expected to be a row vector of length n, with the whole
// set stored as an nv x n matrix w in column major order :(
// to get at the jth vector in the set, you must index the elements as
// w + 0 * nv + j, w + nv + j, w + 2 * nv + j, ..., w + (n - 1) * nv + j

extern "C"
void gpu2MahalanobisHelper(const int * vectorSetSize, const int * vectorLength,
	const double * vectorSet1, const double * vectorSet2, const double * mat,
	double * result)
{
	int
		nv = *vectorSetSize, n = *vectorLength;

	double
		* dVectorSet1, * dVectorSet2, * dMat,
		* temp1, * temp2, * dResult;

	size_t
		resultBytes = nv * sizeof(double),
		matBytes = n * n * sizeof(double),
		vectorSetBytes = nv * n * sizeof(double);

	hipMalloc((void **) & dVectorSet1, vectorSetBytes);
	hipMalloc((void **) & dVectorSet2, vectorSetBytes);
	hipMalloc((void **) & dMat, matBytes);
	hipMalloc((void **) & temp1, vectorSetBytes);
	hipMalloc((void **) & temp2, vectorSetBytes);
	hipMalloc((void **) & dResult, resultBytes);

	hipMemcpy(dVectorSet1, vectorSet1, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dVectorSet2, vectorSet2, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dMat, mat, matBytes, hipMemcpyHostToDevice);

	dim3 dimBlock(NTHREADS, 1, 1);
	int gx = ceil((double) nv /(double) dimBlock.x);
	dim3 dimGrid(gx, 1, 1);

	hipDeviceSynchronize();
	ddMahalanobisHelper<<<dimBlock, dimGrid>>>(nv, n, dVectorSet1, dVectorSet2,
		dMat, temp1, temp2, dResult);
	hipDeviceSynchronize();

	hipFree(temp1);
	hipFree(temp2);
	hipFree(dVectorSet1);
	hipFree(dVectorSet2);
	hipFree(dMat);

	hipMemcpy(result, dResult, resultBytes, hipMemcpyDeviceToHost);
	hipFree(dResult);
}

__global__ void dMahaDouble(int nvectors, int n,
	const double * vectors1, const double * vectors2, const double * mat,
	double * result)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= nvectors) return;

	int
		k, col, innerCol;
	double
		sum = 0, innerSum;

	for(int j = 0; j < n; j++) {
		innerSum = 0;
		col = j * n;
		for(k = 0; k < n; k++) {
			innerCol = i + k * nvectors;
			innerSum += (vectors1[innerCol] - vectors2[innerCol])
				* mat[k + col];
		}
		col = i + j * nvectors;
		sum += innerSum * (vectors1[col] - vectors2[col]);
	}
	result[i] = sqrtf(sum);
}

extern "C"
void gpuMahaDouble(const int * vectorSetSize, const int * vectorLength,
	const double * vectorSet1, const double * vectorSet2, const double * mat,
	double * result)
{
	int
		nv = *vectorSetSize, n = *vectorLength;

	double
		* dVectorSet1, * dVectorSet2, * dMat, * dResult;

	size_t
		resultBytes = nv * sizeof(double),
		matBytes = n * n * sizeof(double),
		vectorSetBytes = nv * n * sizeof(double);

	hipMalloc((void **) & dVectorSet1, vectorSetBytes);
	hipMalloc((void **) & dVectorSet2, vectorSetBytes);
	hipMalloc((void **) & dMat, matBytes);
	hipMalloc((void **) & dResult, resultBytes);

	hipMemcpy(dVectorSet1, vectorSet1, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dVectorSet2, vectorSet2, vectorSetBytes, hipMemcpyHostToDevice);
	hipMemcpy(dMat, mat, matBytes, hipMemcpyHostToDevice);

	dim3 dimBlock(NTHREADS, 1, 1);
	int gx = ceil((double) nv /(double) dimBlock.x);
	dim3 dimGrid(gx, 1, 1);

	hipDeviceSynchronize();
	dMahaDouble<<<dimBlock, dimGrid>>>(nv, n, dVectorSet1, dVectorSet2, dMat,
		dResult);
	hipDeviceSynchronize();

	hipMemcpy(result, dResult, resultBytes, hipMemcpyDeviceToHost);

	hipFree(dVectorSet1);
	hipFree(dVectorSet2);
	hipFree(dMat);
	hipFree(dResult);
}
